#include "hip/hip_runtime.h"

#ifndef _FILTER_KERNEL_H_
#define _FILTER_KERNEL_H_

__global__ void SobelFilter(unsigned char* g_DataIn, unsigned char* g_DataOut, int width, int height)
{
   __shared__ unsigned char sharedMem[BLOCK_HEIGHT * BLOCK_WIDTH];

   int x = blockIdx.x * TILE_WIDTH + threadIdx.x ;//- FILTER_RADIUS;
   int y = blockIdx.y * TILE_HEIGHT + threadIdx.y ;//- FILTER_RADIUS;


   int index = y * (width) + x;

   // STUDENT:  Check 1


   if (x >= width || y >= height)
      return;

   // STUDENT: Check 2

   if( x < FILTER_RADIUS || y < FILTER_RADIUS) {
       g_DataOut[index] = g_DataIn[index];
       return;
    }

   if ((x > width - FILTER_RADIUS - 1)&&(x <width)) {
       g_DataOut[index] = g_DataIn[index];
       return;
    }

    if ((y > height - FILTER_RADIUS - 1)&&(y < height)) {
       g_DataOut[index] = g_DataIn[index];
       return;
    }


   int sharedIndex = threadIdx.y * blockDim.y + threadIdx.x;
   sharedMem[sharedIndex] = g_DataIn[index];
   __syncthreads();


   //STUDENT: Make sure only the thread ids should write the sum of the neighbors.
   const float SobelMatrix[9] = {-1,0,1,-2,0,2,-1,0,1};
   float sumX = 0, sumY=0;

   // Adding the code that performs Sobel filter operation
   
   float pixelIndex = 0.0;
   for(int destY = -FILTER_RADIUS; destY <= FILTER_RADIUS; destY++) {
      for(int destX = -FILTER_RADIUS; destX <= FILTER_RADIUS; destX++) {
             pixelIndex = (float)(g_DataIn[y*width + x +  (destY * width + destX)]);
             sumX += pixelIndex * SobelMatrix[(destY + FILTER_RADIUS) * FILTER_DIAMETER + (destX+FILTER_RADIUS)];
             sumY += pixelIndex * SobelMatrix[(destX + FILTER_RADIUS) * FILTER_DIAMETER + (destY+FILTER_RADIUS)];
	}
   }

   // computing the g_DataOut
   g_DataOut[index] = abs(sumX) + abs(sumY) > EDGE_VALUE_THRESHOLD ? 255 : 0;
}

__global__ void AverageFilter(unsigned char* g_DataIn, unsigned char* g_DataOut, int width, int height)
{
    __shared__ unsigned char sharedMem[BLOCK_HEIGHT*BLOCK_WIDTH];

   int x = blockIdx.x * TILE_WIDTH + threadIdx.x ;//- FILTER_RADIUS;
   int y = blockIdx.y * TILE_HEIGHT + threadIdx.y ;//- FILTER_RADIUS;

   // Get the Global index into the original image
          int index = y * (width) + x;
	  
	  const float SobelMatrix[9] = {1,1,1,1,1,1,1,1,1};
          float sumX = 0;

 if (x >= width || y >= height)
      return;

   // STUDENT: Check 2
   // Handle the border cases of the global image
   if( x < FILTER_RADIUS || y < FILTER_RADIUS) {
       g_DataOut[index] = g_DataIn[index];
       return;
   }

   if ((x > width - FILTER_RADIUS - 1)&&(x <width)) {
       g_DataOut[index] = g_DataIn[index];
       return;
   }

   if ((y > height - FILTER_RADIUS - 1)&&(y < height)) {
       g_DataOut[index] = g_DataIn[index];
       return;
   }

               
    // STUDENT: write code for Average Filter : use Sobel as base code
    for(int dy = -FILTER_RADIUS; dy <= FILTER_RADIUS; dy++) {
        for(int dx = -FILTER_RADIUS; dx <= FILTER_RADIUS; dx++) {
             float Pixel = (float)(g_DataIn[y*width + x +  (dy * width + dx)]);
             sumX += Pixel * SobelMatrix[(dy + FILTER_RADIUS) * FILTER_DIAMETER + (dx+FILTER_RADIUS)];
        }
    }

    g_DataOut[index] = (sumX)/FILTER_AREA ;

}



__global__ void HighBoostFilter(unsigned char* g_DataIn, unsigned char* g_DataOut, int width, int height)
{
  __shared__ unsigned char sharedMem[BLOCK_HEIGHT*BLOCK_WIDTH];

  int x = blockIdx.x * TILE_WIDTH + threadIdx.x ;//- FILTER_RADIUS;
  int y = blockIdx.y * TILE_HEIGHT + threadIdx.y ;//- FILTER_RADIUS;

  // Get the Global index into the original image
  int index = y * (width) + x;
  const float SobelMatrix[9] = {1,1,1,1,1,1,1,1,1};
  float sumX = 0, sumY = 0, Pixel = 0 ;

 if (x >= width || y >= height)
      return;

   // STUDENT: Check 2
   // Handle the border cases of the global image
   if( x < FILTER_RADIUS || y < FILTER_RADIUS) {
       g_DataOut[index] = g_DataIn[index];
       return;
    }

   if ((x > width - FILTER_RADIUS - 1)&&(x <width)) {
       g_DataOut[index] = g_DataIn[index];
       return;
    }

    if ((y > height - FILTER_RADIUS - 1)&&(y < height)) {
       g_DataOut[index] = g_DataIn[index];
       return;
    }

    int sharedIndex = threadIdx.y * blockDim.y + threadIdx.x;
    sharedMem[sharedIndex] = g_DataIn[index];
    __syncthreads();


    // STUDENT: write code for High Boost Filter : use Sobel as base code
    for(int dy = -FILTER_RADIUS; dy <= FILTER_RADIUS; dy++) {
        for(int dx = -FILTER_RADIUS; dx <= FILTER_RADIUS; dx++) {
            Pixel = (float)(g_DataIn[y*width + x +  (dy * width + dx)]);
            sumX += Pixel * SobelMatrix[(dy + FILTER_RADIUS) * FILTER_DIAMETER + (dx+FILTER_RADIUS)];
 
        }
    }

    g_DataOut[index] = CLAMP_8bit((int)(Pixel + HIGH_BOOST_FACTOR * (unsigned char)(Pixel - sumX / FILTER_AREA)));

}


#endif // _FILTER_KERNEL_H_


